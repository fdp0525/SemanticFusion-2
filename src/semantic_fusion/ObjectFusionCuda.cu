#include <stdio.h>
#include <assert.h> 

#include <hip/hip_runtime.h>

#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool
        abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n",
                hipGetErrorString(code), file, line);
        if (abort) exit(code);
    } 
}


__global__ 
void updateTable(int num_to_update, const int* deleted_ids, const int num_deleted, const int current_table_size,
                 float const* object_id_table, const int prob_width, const int prob_height, 
                 const int new_prob_width, float* new_object_id_table)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;  // kernal index
    if (index < num_to_update) {
        const int channel_id = index / new_prob_width;  // get class id of current kernal in new table
        const int component_id = index - (channel_id * new_prob_width);  // get surfel id of current kernal in new table
        // const int new_id = (class_id * prob_width) + component_id; // get table index with max_componets as width
        if (component_id >= num_deleted) {
            // Initialise to prior (prob height is the number of classes)
            // new_object_id_table[new_id] = 1.0f / prob_height;
            // Reset the max class surfel colouring lookup
            new_object_id_table[component_id] = 0.0;  // obj id
            new_object_id_table[component_id + prob_width] = 1.0; // obj conf
            new_object_id_table[component_id + prob_width + prob_width] = 0.0;
        } else {
            int offset = deleted_ids[component_id]; // get corresponded surf_id in previous table
            // new_object_id_table[new_id] = object_id_table[(class_id * prob_width) + offset];
            // Also must update our max class mapping
            new_object_id_table[component_id] = object_id_table[offset];
            new_object_id_table[component_id + prob_width] = object_id_table[prob_width + offset];
            new_object_id_table[component_id + prob_width + prob_width] = object_id_table[prob_width + prob_width + offset];
        }
    }
}



__host__
void updateObjectTable(int* deleted_ids, const int num_deleted, const int current_table_size,
                            float const* object_id_table, const int prob_width, const int prob_height, 
                          const int new_table_width, float* new_object_id_table){
    const int threads = 512;
    const int num_to_update = new_table_width;// * prob_height; // new_table_width*num_classes_
    const int blocks = (num_to_update + threads - 1) / threads;  
    dim3 dimGrid(blocks);
    dim3 dimBlock(threads);
    updateTable<<<dimGrid,dimBlock>>>(num_to_update,deleted_ids,num_deleted,current_table_size,
    									object_id_table,prob_width,prob_height,
    									new_table_width,new_object_id_table);
    gpuErrChk(hipGetLastError());
    gpuErrChk(hipDeviceSynchronize());



}


__global__ 
void objectTableUpdate(hipTextureObject_t ids, const int ids_width, const int ids_height, 
                          const float* mask_probabilities,
                    const int x1, const int y1, const int box_width, const int box_height, 
                    const int obj_id, const int class_id, const float class_prob,
                    float* object_id_table, const int map_size)
{
	// masks coordinate indices 
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    // New uniqueness code
    if (x<box_width||y<box_height){
    // if (1){
		const int check_patch = 16;
		const int x_min = (x - check_patch) < 0 ? 0 : (x - check_patch);
		const int x_max = (x + check_patch) > box_width ? box_width : (x + check_patch);
		const int y_min = (y - check_patch) < 0 ? 0 : (y - check_patch);
		const int y_max = (y + check_patch) < box_height ? box_height : (y + check_patch);
    	int surfel_id = tex2D<int>(ids,x+x1,y+y1);
    	// int surfel_id = 0;
    	int first_h, first_w;

	    for (int h = y_min; h < y_max; ++h) {
        int other_surfel_id;
        for (int w = x_min; w < x_max; ++w) {
            other_surfel_id = tex2D<int>(ids,w+x1,h+y1);
            if (other_surfel_id == surfel_id) {
                first_h = h;
                first_w = w;
                break;
            }
        }
        if (other_surfel_id == surfel_id) {
                break;
            }
    	}

	    if (first_h != y || first_w != x) {
	        surfel_id = 0;
	    }

    	if (surfel_id>0) {
    		// pointer at (x,y) on mask
        	const float* probability = mask_probabilities + (y * box_width + x);
        	// pointer at surfel_id on table
	        float* prior_id = object_id_table + surfel_id;

	        // TO DO: fusion
	        //	    
	        if(mask_probabilities[y*box_width+x] > 0.4){
		        object_id_table[surfel_id] = static_cast<float>(obj_id);
	   			object_id_table[surfel_id + map_size] = 1.0;	
	        	object_id_table[surfel_id + map_size + map_size] += 1.0;
	        }    
    	} 
    		

    }
    
}



__host__
void fuseObjects(hipTextureObject_t ids, const int ids_width, const int ids_height, const float* mask_probabilities,
                    const int x1, const int y1, const int box_width, const int box_height, const int obj_id, const int class_id, const float class_prob,
                    float* object_id_table, const int map_size){
	// NOTE Res must be pow 2 and > 32
    const int blocks = 32; // TODO : global function need check
    dim3 dimGrid(blocks,blocks);
    dim3 dimBlock((box_width+blocks-1)/blocks,(box_height+blocks-1)/blocks);
    objectTableUpdate<<<dimGrid,dimBlock>>>(ids,ids_width,ids_height,mask_probabilities,
    	x1,y1,box_width,box_height, obj_id, class_id, class_prob, object_id_table, map_size);
    gpuErrChk(hipGetLastError());
    gpuErrChk(hipDeviceSynchronize());
}


__global__ 
void renderObjectMapKernel(hipTextureObject_t ids, const int ids_width, const int ids_height, 
                          const float* object_id_table, const int prob_width, const int prob_height, 
                          float* rendered_objects) 
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int surfel_id = tex2D<int>(ids,x,y);
    int projected_object_offset = y * ids_width + x;
    int object_table_offset = surfel_id;
    if (surfel_id > 0) {
        rendered_objects[projected_object_offset] = object_id_table[object_table_offset]+1;
    } else {
        rendered_objects[projected_object_offset] = 0.0; // ((class_id == 0) ? 1.0 : 0.0);
    }
    // object_table_offset += prob_width;
}
__host__
void renderObjectMap(hipTextureObject_t ids, const int ids_width, const int ids_height, 
                          const float* object_id_table, const int prob_width, const int prob_height, 
                          float* rendered_objects)
{
    // NOTE Res must be pow 2 and > 32
    const int blocks = 32; // TODO : global function need check
    dim3 dimGrid(blocks,blocks);
    // dim3 dimBlock(ids_width/blocks,ids_height/blocks);
    dim3 dimBlock((ids_width+blocks-1)/blocks,(ids_height+blocks-1)/blocks);
    renderObjectMapKernel<<<dimGrid,dimBlock>>>(ids,ids_width,ids_height,object_id_table,prob_width,prob_height,rendered_objects);
    gpuErrChk(hipGetLastError());
    gpuErrChk(hipDeviceSynchronize());
}